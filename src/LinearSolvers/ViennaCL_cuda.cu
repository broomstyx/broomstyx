#include "hip/hip_runtime.h"
/*
  Copyright (c) 2014 - 2019 University of Bergen
  
  This file is part of the BROOMStyx project.

  BROOMStyx is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  BROOMStyx is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with BROOMStyx.  If not, see <http://www.gnu.org/licenses/>.

  Consult the COPYING file in the top-level source directory of this
  module for the precise wording of the license and the AUTHORS file
  for the list of copyright holders.
*/

#include "ViennaCL_cuda.hpp"

#ifdef HAVE_VIENNACL

#include "Core/ObjectFactory.hpp"
#include "Util/readOperations.hpp"
#include "Util/RealVector.hpp"
#include "SparseMatrix/SparseMatrix.hpp"

#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"
#include "viennacl/compressed_matrix.hpp"
#include "viennacl/linalg/prod.hpp"
#include "viennacl/linalg/ilu_operations.hpp"
#include "viennacl/linalg/ilu.hpp"
#include "viennacl/linalg/bicgstab.hpp"
#include "viennacl/linalg/gmres.hpp"

void copy_csr0_matrix( broomstyx::SparseMatrix* coefMat, viennacl::compressed_matrix<double>& gpuMatrix )
{
    int* csr_rows;
    int* csr_cols;
    double * csr_elements;
    int num_rows, num_cols, num_nnz;
    
    std::tie(csr_rows, csr_cols) = coefMat->giveProfileArrays();
    csr_elements = coefMat->giveValArray();
    std::tie(num_rows, num_cols) = coefMat->giveMatrixDimensions();
    num_nnz = coefMat->giveNumberOfNonzeros();
    
    if ( num_rows > 0 && num_cols > 0 && num_nnz > 0)
    {
        viennacl::backend::typesafe_host_array<unsigned int> row_buffer(gpuMatrix.handle1(), num_rows + 1);

        if (sizeof(int) != row_buffer.element_size()) // check whether indices are of the same length (same number of bits)
        {
            viennacl::backend::typesafe_host_array<unsigned int> col_buffer(gpuMatrix.handle2(), num_nnz);

            for ( int i=0; i<=num_rows; ++i )
                row_buffer.set( i, csr_rows[i] );
            for ( int i=0; i<num_nnz; ++i )
                col_buffer.set(i, csr_cols[i]);

            gpuMatrix.set(row_buffer.get(), col_buffer.get(), csr_elements, num_rows, num_cols, num_nnz);
        }
        else
            gpuMatrix.set(static_cast<const void*>(csr_rows), static_cast<const void*>(csr_cols), csr_elements, num_rows, num_cols, num_nnz);
    }
}

using namespace broomstyx;

registerBroomstyxObject(LinearSolver, ViennaCL_cuda)

// Constructor
ViennaCL_cuda::ViennaCL_cuda() : LinearSolver() {}

// Destructor
ViennaCL_cuda::~ViennaCL_cuda() {}

// Public methods
// ---------------------------------------------------------------------------
std::string ViennaCL_cuda::giveRequiredMatrixFormat()
{
    return std::string("CSR0");
}
// ---------------------------------------------------------------------------
void ViennaCL_cuda::readDataFrom( FILE* fp )
{
    std::string src = "ViennaCL_cuda (LinearSolver)";
    
    verifyKeyword(fp, "Algorithm", src);
    _algorithm = getStringInputFrom(fp, "Failed to read iterative algorithm for linear solver from input file!", src);
    
    _tol = getRealInputFrom(fp, "Failed to read relative tolerance for iterative linear solver from input file!", src);
    _maxIter = getIntegerInputFrom(fp, "Failed to read max. iterations for iterative linear solver from input file!", src);
    
    if ( _algorithm == "GMRES" )
        _restart = getIntegerInputFrom(fp, "Failed to read number of iterations before restarting GMRES solver from input file!", src);
    
    verifyKeyword(fp, "Preconditioner", src);
    _preconditioner = getStringInputFrom(fp, "Failed to read preconditioner for linear solver from input file!", src);
    
    if ( _preconditioner == "Chow_Patel_ILU0" )
    {
        _chowPatel_sweep = getIntegerInputFrom(fp, "Failed to read number of sweeps for preconditioner Chow_Patel_ILU0 from input file!", src);
        _chowPatel_nJacIter = getIntegerInputFrom(fp, "Failed to read number of Jacobi iterations for preconditioner Chow_Patel_ILU0 from input file!", src);
    }
    else if ( _preconditioner == "ILU0" || _preconditioner == "none" )
    {
        // Do nothing.
    }
    else
    {
        std::string errMsg = "Invalid preconditioner tag '" + _preconditioner + "' encountered while reading input file!\nSource: ViennaCL_cuda (LinearSolver)";
        throw std::runtime_error(errMsg);
    }
}
// ---------------------------------------------------------------------------
void ViennaCL_cuda::setInitialGuessTo( RealVector& initGuess )
{
//    // Temporary hack: last solution becomes initial guess :)
//    if ( _initGuess.dim() == 0 )
        _initGuess = initGuess;
}
// ---------------------------------------------------------------------------
RealVector ViennaCL_cuda::solve( SparseMatrix* coefMat, RealVector& rhs )
{
    // Instantiate objects
    int nUnknowns = rhs.dim();
    
    RealVector soln(nUnknowns);
    viennacl::compressed_matrix<double> gpuMatrix;
    viennacl::vector<double> gpuRhs(nUnknowns);
    viennacl::vector<double> gpuResult(nUnknowns);
    viennacl::vector<double> gpuInitGuess(nUnknowns);
    
    // Copy data from CPU into GPU
    copy_csr0_matrix(coefMat, gpuMatrix);
    viennacl::copy(rhs.ptr(), rhs.ptr() + nUnknowns, gpuRhs.begin());
    viennacl::copy(_initGuess.ptr(), _initGuess.ptr() + nUnknowns, gpuInitGuess.begin());
    
    // Setup iterative solver
    if ( _algorithm == "BiCGStab" )
    {
        viennacl::linalg::bicgstab_tag solverTag(_tol, _maxIter);
        viennacl::linalg::bicgstab_solver<viennacl::vector<double> > iterSolver(solverTag);
        iterSolver.set_initial_guess(gpuInitGuess);
        
        // Setup preconditioner
        if ( _preconditioner == "Chow_Patel_ILU0" )
        {
            viennacl::linalg::chow_patel_tag pcConfig;
            pcConfig.sweeps(_chowPatel_sweep);
            pcConfig.jacobi_iters(_chowPatel_nJacIter);
            viennacl::linalg::chow_patel_ilu_precond< viennacl::compressed_matrix<double> > pcObject(gpuMatrix, pcConfig);
            std::printf("\n      Preconditioner setup completed.");
            std::fflush(stdout);
            gpuResult = iterSolver(gpuMatrix, gpuRhs, pcObject);
        }
        else if ( _preconditioner == "ILU0" )
        {
            viennacl::linalg::ilu0_tag pcConfig;
            viennacl::linalg::ilu0_precond< viennacl::compressed_matrix<double> > pcObject(gpuMatrix, pcConfig);
            std::printf("\n      Preconditioner setup completed.");
            std::fflush(stdout);
            gpuResult = iterSolver(gpuMatrix, gpuRhs, pcObject);
        }
        else if ( _preconditioner == "none" )
        {
            gpuResult = iterSolver(gpuMatrix, gpuRhs);
        }
        else
        {
            std::string errmsg = "Preconditioner '" + _preconditioner + "' is not yet programmed!\n";
            throw std::runtime_error(errmsg);
        }
        
        std::printf("\n      System solved.");
        std::printf("\n      Num iters = %d, est. error = %e\n", (int)iterSolver.tag().iters(), iterSolver.tag().error());
    }
    else if ( _algorithm == "GMRES" )
    {
        viennacl::linalg::gmres_tag solverTag(_tol, _maxIter, _restart);
        viennacl::linalg::gmres_solver<viennacl::vector<double> > iterSolver(solverTag);
        iterSolver.set_initial_guess(gpuInitGuess);
        
        // Setup preconditioner
        if ( _preconditioner == "Chow_Patel_ILU0" )
        {
            viennacl::linalg::chow_patel_tag pcConfig;
            pcConfig.sweeps(_chowPatel_sweep);
            pcConfig.jacobi_iters(_chowPatel_nJacIter);
            viennacl::linalg::chow_patel_ilu_precond< viennacl::compressed_matrix<double> > pcObject(gpuMatrix, pcConfig);
            std::printf("\n      Preconditioner setup completed.");
            std::fflush(stdout);
            gpuResult = iterSolver(gpuMatrix, gpuRhs, pcObject);
        }
        else if ( _preconditioner == "ILU0" )
        {
            viennacl::linalg::ilu0_tag pcConfig;
            viennacl::linalg::ilu0_precond< viennacl::compressed_matrix<double> > pcObject(gpuMatrix, pcConfig);
            std::printf("\n      Preconditioner setup completed.");
            std::fflush(stdout);
            gpuResult = iterSolver(gpuMatrix, gpuRhs, pcObject);
        }
        else if ( _preconditioner == "none" )
        {
            gpuResult = iterSolver(gpuMatrix, gpuRhs);
        }
        else
        {
            std::string errmsg = "Preconditioner '" + _preconditioner + "' is not yet programmed!\n";
            throw std::runtime_error(errmsg);
        }
        
        std::printf("\n      System solved.");
        std::printf("\n      Num iters = %d, est. error = %e\n", (int)iterSolver.tag().iters(), iterSolver.tag().error());
    }
    else
    {
        std::string errmsg = "Iterative algorithm '" + _algorithm + "' is not yet programmed!\n";
        throw std::runtime_error(errmsg);
    }
    
    // Copy result vector from GPU into CPU
    viennacl::copy(gpuResult.begin(), gpuResult.end(), soln.ptr());
    
//    // Copy to initial guess (just a hack at the moment)
//    _initGuess = soln;
    
//    std::printf("    %-40s", "");
    
    return soln;
}
// ---------------------------------------------------------------------------
bool ViennaCL_cuda::takesInitialGuess()
{
    return true;
}

#endif /* HAVE_VIENNACL */
